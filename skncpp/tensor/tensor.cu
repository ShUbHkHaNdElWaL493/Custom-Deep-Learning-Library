#include "hip/hip_runtime.h"
/*
    Shubh Khandelwal
*/

#include <hip/hip_runtime.h>
#include <iostream>
#include "tensor.hpp"

__global__ void kernel_scalar_add(float* A, float scalar, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size)
    {
        A[i] += scalar;
    }
}

__global__ void kernel_add(const float* A, const float* B, float* out, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size)
    {
        out[i] = A[i] + B[i];
    }
}

__global__ void kernel_multiply(const float* A, const float* B, float* out, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size)
    {
        out[i] = A[i] * B[i];
    }
}

__global__ void kernel_matrix_multiply(const float* A, const float* B, float* out, int C, int I, int K, int J)
{

    int c = blockIdx.z;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < I && j < J)
    {
        float sum = 0;
        for (int k = 0; k < K; ++k)
        {
            int idxA = (c * I + i) * K + k;
            int idxB = (c * K + k) * J + j;
            sum += A[idxA] * B[idxB];
        }
        int idxOut = (c * I * J) + (i * J) + j;
        out[idxOut] = sum;
    }

}

Tensor::Tensor()
{}

Tensor::Tensor(int channels, int rows, int columns, Device device) : device(device)
{

    shape[0] = channels;
    shape[1] = rows;
    shape[2] = columns;

    size = 1;
    for (int dim : shape)
    {
        size *= dim;
    }

    if (device == Device::CPU)
    {
        data = new float[size];
    } else if (device == Device::GPU)
    {
        CUDA_CHECK(hipMalloc(&data, size * sizeof(float)));
    } else
    {
        std::cerr << "UNKNOWN_DEVICE_ERROR" << std::endl;
        exit(1);
    }

}

Tensor::~Tensor()
{
    if (device == Device::CPU)
    {
        delete[] data;
    } else if (device == Device::GPU)
    {
        CUDA_CHECK(hipFree(data));
    } else
    {
        std::cerr << "UNKNOWN_DEVICE_ERROR" << std::endl;
        exit(1);
    }
}

const Device Tensor::get_device() const
{
    return device;
}

const int* Tensor::get_shape() const
{
    return shape;
}

void Tensor::to(Device device)
{

    if (this->device == device)
    {
        return;
    }

    if (device == Device::CPU)
    {
        float *temp = new float[size];
        CUDA_CHECK(hipMemcpy((void*) temp, (void*) data, size * sizeof(float), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipFree(data));
        data = temp;
    } else if (device == Device::GPU)
    {
        float *temp;
        CUDA_CHECK(hipMalloc(&temp, size * sizeof(float)));
        CUDA_CHECK(hipMemcpy((void*) temp, (void*) data, size * sizeof(float), hipMemcpyHostToDevice));
        delete[] data;
        data = temp;
    } else
    {
        std::cerr << "UNKNOWN_DEVICE_ERROR" << std::endl;
        exit(1);
    }

    this->device = device;

}

Tensor Tensor::copy() const
{
    Tensor clone(shape[0], shape[1], shape[2], device);
    if (device == Device::CPU)
    {
        std::copy(data, data + size, clone.data);
    } else if (device == Device::GPU)
    {
        CUDA_CHECK(hipMemcpy(clone.data, data, size * sizeof(float), hipMemcpyDeviceToDevice));
    } else
    {
        std::cerr << "UNKNOWN_DEVICE_ERROR" << std::endl;
        exit(1);
    }
    return clone;
}

void Tensor::reshape(int channels, int rows, int columns)
{
    size_t new_size = channels * rows * columns;
    if (new_size != size)
    {
        std::cerr << "DIMENSION_MISMATCH_ERROR" << std::endl;
        exit(4);
    }
    shape[0] = channels;
    shape[1] = rows;
    shape[2] = columns;
}

size_t Tensor::index(int channel, int row, int column) const
{

    if (channel >= shape[0] || row >= shape[1] || column >= shape[2])
    {
        std::cerr << "DIMENSION_MISMATCH_ERROR" << std::endl;
        exit(4);
    }

    return ((channel * shape[1] + row) * shape[2] + column);

}

float Tensor::get(int channel, int row, int column) const
{
    if (device != Device::CPU)
    {
        std::cerr << "ILLEGAL_DEVICE_ERROR" << std::endl;
        exit(2);
    }
    return data[index(channel, row, column)];
}

void Tensor::set(int channel, int row, int column, float value)
{
    if (device != Device::CPU)
    {
        std::cerr << "ILLEGAL_DEVICE_ERROR" << std::endl;
        exit(2);
    }
    data[index(channel, row, column)] = value;
}

void Tensor::zeros()
{
    if (device == Device::CPU)
    {
        std::fill(data, data + size, 0.0f);
    } else if (device == Device::GPU)
    {
        CUDA_CHECK(hipMemset(data, 0, size * sizeof(float)));
    } else
    {
        std::cerr << "UNKNOWN_DEVICE_ERROR" << std::endl;
        exit(1);
    }
}

void Tensor::scalar_add(float scalar)
{
    if (device == Device::CPU)
    {
        for (size_t i = 0; i < size; i++)
        {
            data[i] += scalar;
        }
    } else if (device == Device::GPU)
    {
        int threads = 256;
        int blocks = (size + threads - 1) / threads;
        kernel_scalar_add<<<blocks, threads>>>(data, scalar, size);
        CUDA_CHECK(hipDeviceSynchronize());
    } else
    {
        std::cerr << "UNKNOWN_DEVICE_ERROR" << std::endl;
        exit(1);
    }
}

void Tensor::add(const Tensor& temp1, const Tensor& temp2)
{

    if ((device != temp1.device) || (device != temp2.device))
    {
        std::cerr << "DEVICE_MISMATCH_ERROR" << std::endl;
        exit(3);
    }

    if ((shape[0] != temp1.shape[0]) || (shape[0] != temp2.shape[0]) \
    || (shape[1] != temp1.shape[1]) || (shape[1] != temp2.shape[1]) \
    || (shape[2] != temp1.shape[2]) || (shape[2] != temp2.shape[2]))
    {
        std::cerr << "DIMENSION_MISMATCH_ERROR" << std::endl;
        exit(4);
    }

    if (device == Device::CPU)
    {
        for (size_t i = 0; i < size; i++)
        {
            data[i] = temp1.data[i] + temp2.data[i];
        }
    } else if (device == Device::GPU)
    {
        int threads = 256;
        int blocks = (size + threads - 1) / threads;
        kernel_add<<<blocks, threads>>>(temp1.data, temp2.data, data, size);
        CUDA_CHECK(hipDeviceSynchronize());
    } else
    {
        std::cerr << "UNKNOWN_DEVICE_ERROR" << std::endl;
        exit(1);
    }
    
}

void Tensor::multiply(const Tensor& temp1, const Tensor& temp2)
{

    if ((device != temp1.device) || (device != temp2.device))
    {
        std::cerr << "DEVICE_MISMATCH_ERROR" << std::endl;
        exit(3);
    }

    if ((shape[0] != temp1.shape[0]) || (shape[0] != temp2.shape[0]) \
    || (shape[1] != temp1.shape[1]) || (shape[1] != temp2.shape[1]) \
    || (shape[2] != temp1.shape[2]) || (shape[2] != temp2.shape[2]))
    {
        std::cerr << "DIMENSION_MISMATCH_ERROR" << std::endl;
        exit(4);
    }

    if (device == Device::CPU)
    {
        for (size_t i = 0; i < size; i++)
        {
            data[i] = temp1.data[i] * temp2.data[i];
        }
    } else if (device == Device::GPU)
    {
        int threads = 256;
        int blocks = (size + threads - 1) / threads;
        kernel_multiply<<<blocks, threads>>>(temp1.data, temp2.data, data, size);
        CUDA_CHECK(hipDeviceSynchronize());
    } else
    {
        std::cerr << "UNKNOWN_DEVICE_ERROR" << std::endl;
        exit(1);
    }

}

void Tensor::matrix_multiply(const Tensor& temp1, const Tensor& temp2)
{

    if ((device != temp1.device) || (device != temp2.device))
    {
        std::cerr << "DEVICE_MISMATCH_ERROR" << std::endl;
        exit(3);
    }

    if ((shape[0] != temp1.shape[0]) \
    || (shape[0] != temp2.shape[0]) \
    || (shape[1] != temp1.shape[1]) \
    || (shape[2] != temp2.shape[2]) \
    || (temp1.shape[2] != temp2.shape[1]))
    {
        std::cerr << "DIMENSION_MISMATCH_ERROR" << std::endl;
        exit(4);
    }

    int C = shape[0];
    int I = shape[1];
    int J = shape[2];
    int K = temp1.shape[2];

    if (device == Device::CPU)
    {
        for (int c = 0; c < shape[0]; c++)
        {
            for (int i = 0; i < shape[1]; i++)
            {
                for (int j = 0; j < shape[2]; j++)
                {
                    float sum = 0;
                    for (int k = 0; k < temp1.shape[2]; k++)
                    {
                        sum += temp1.get(c, i, k) * temp2.get(c, k, j);
                    }
                    set(c, i, j, sum);
                }
            }
        }
    } else if (device == Device::GPU)
    {
        dim3 blockDim(16, 16);
        dim3 gridDim((J + blockDim.x - 1) / blockDim.x, (I + blockDim.y - 1) / blockDim.y, C);
        kernel_matrix_multiply<<<gridDim, blockDim>>>(temp1.data, temp2.data, data, C, I, K, J);
        CUDA_CHECK(hipDeviceSynchronize());
    } else
    {
        std::cerr << "UNKNOWN_DEVICE_ERROR" << std::endl;
        exit(1);
    }

}